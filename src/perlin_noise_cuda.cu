#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// Constants
#define WIDTH 10000
#define HEIGHT 10000
#define CHANNELS 3

// Interpolation function
__device__ float lerp(float a, float b, float t)
{
    return a + (b - a) * t;
}

// Smoothstep function
__device__ float smoothstep(float t)
{
    return t * t * (3 - 2 * t);
}

// Generate random gradient vectors
__global__ void generate_gradients(float *grad_x, float *grad_y, int width, int height, unsigned long seed)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height)
    {
        hiprandState state;
        hiprand_init(seed * (x * height + y), 0, 0, &state);
        int index = y * width + x;
        float angle = hiprand_uniform(&state) * 2 * M_PI;
        grad_x[index] = cos(angle);
        grad_y[index] = sin(angle);
    }
}

// Perlin noise function
__device__ float perlin(float x, float y, float *grad_x, float *grad_y, int width, int height)
{
    int ix = (int)floor(x);
    int iy = (int)floor(y);
    float fx = x - ix;
    float fy = y - iy;

    ix = ix % width;
    iy = iy % height;

    int ix1 = (ix + 1) % width;
    int iy1 = (iy + 1) % height;

    float dot00 = grad_x[iy * width + ix] * fx + grad_y[iy * width + ix] * fy;
    float dot10 = grad_x[iy * width + ix1] * (fx - 1) + grad_y[iy * width + ix1] * fy;
    float dot01 = grad_x[iy1 * width + ix] * fx + grad_y[iy1 * width + ix] * (fy - 1);
    float dot11 = grad_x[iy1 * width + ix1] * (fx - 1) + grad_y[iy1 * width + ix1] * (fy - 1);

    float sx = smoothstep(fx);
    float sy = smoothstep(fy);

    float a = lerp(dot00, dot10, sx);
    float b = lerp(dot01, dot11, sx);

    return lerp(a, b, sy);
}

__global__ void render_image(unsigned char *data, float *grad_x, float *grad_y, int width, int height, float scale)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        float value = perlin(x / scale, y / scale, grad_x, grad_y, width, height);

        value = (value + 1) / 2.0;
        unsigned char color = (unsigned char)(value * 255);

        int index = (y * width + x) * CHANNELS;
        data[index] = color;
        data[index + 1] = color;
        data[index + 2] = color;
    }
}

int main(int argc, char *argv[])
{
    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate the data array on the device
    unsigned char *d_data;
    hipMalloc((void **)&d_data, WIDTH * HEIGHT * CHANNELS * sizeof(unsigned char));

    float *d_grad_x;
    hipMalloc((void **)&d_grad_x, WIDTH * HEIGHT * sizeof(float));
    float *d_grad_y;
    hipMalloc((void **)&d_grad_y, WIDTH * HEIGHT * sizeof(float));

    dim3 blockSize(16, 16);
    dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, (HEIGHT + blockSize.y - 1) / blockSize.y);

    hipEventRecord(start, 0);
    generate_gradients<<<gridSize, blockSize>>>(d_grad_x, d_grad_y, WIDTH, HEIGHT, time(NULL));
    hipDeviceSynchronize();

    float scale = 10.0;
    render_image<<<gridSize, blockSize>>>(d_data, d_grad_x, d_grad_y, WIDTH, HEIGHT, scale);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("CUDA time to generate image: %f ms\n", elapsedTime);

    // Copy the data from the device to the host
    unsigned char *h_data = (unsigned char *)malloc(WIDTH * HEIGHT * CHANNELS * sizeof(unsigned char));
    hipMemcpy(h_data, d_data, WIDTH * HEIGHT * CHANNELS * sizeof(unsigned char), hipMemcpyDeviceToHost);

    stbi_write_jpg("out/cuda.jpg", WIDTH, HEIGHT, CHANNELS, h_data, WIDTH * CHANNELS);

    // Free the allocated memory
    free(h_data);
    hipFree(d_data);
    hipFree(d_grad_x);
    hipFree(d_grad_y);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}